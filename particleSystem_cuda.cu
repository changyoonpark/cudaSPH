#include "hip/hip_runtime.h"
#include "particleSystem_cuda.cuh"

void DeviceMemoryPointers::cleanUpDevice(){

	cout << "Device Cleaned Up." << endl;

	hipFree(this->pos);
	hipFree(this->vel);
	hipFree(this->acc);

}

void MemoryOperations::printVector(vector<float4> v){
	for (int i=0;i<v.size();i++){
		cout << v[i].x << "," << v[i].y << "," << v[i].z << endl;
	}
}

void MemoryOperations::foo(vector<float4> v){
	float* foofoo = (float*) v.data();
	cout << "HAHA" << endl;
	for (int i=0;i<v.size();i++){
		cout << foofoo[i] << endl;
	}
	cout << "HAHA" << endl;
}


void MemoryOperations::deviceToHostCopy(
	vector<float4>* hostVector,
	float* devMem){

	hipMemcpy(hostVector->data(),
			   devMem,
			   hostVector->size() * sizeof(float) * 4,
			   hipMemcpyDeviceToHost);

}

void MemoryOperations::hostToDeviceCopy(
	vector<float4>* hostVector,
	float* devMem){
	
	hipMemcpy(devMem,
			   (float*) hostVector->data(),
			   hostVector->size() * sizeof(float) * 4,
			   hipMemcpyHostToDevice);

}

